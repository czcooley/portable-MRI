
#include <hip/hip_runtime.h>



// this version of the kernel supports PE
// this is the pixel dependent field variation from rot to rot expressed in radians


__device__ void multiply_2complex( float a_re,float a_im,
				   float b_re,float b_im,
				   float *c_re,float *c_im )
{
  *c_re = a_re*b_re - a_im*b_im;
  *c_im = a_re*b_im + a_im*b_re;

}




__global__ void forw_mod_fully3D_cudaKernel_v1( float *y_re,float *y_im,
						float *x_re,float *x_im,
						float *times,float *fieldmaps,
						float *b1m_re,float *b1m_im,
						float *b1p_re,float *b1p_im,
						float *b1p_echoe_exp,
						float *freqs,float *freq_weights,
                        float *PE_field,
                        int nfreqs,
						int nsamples,int ncoils,int nrots,
						int nechoes,int npix )
{
  
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  if( index < nsamples*ncoils*nrots*nechoes*nfreqs ){
    
    int tmp1 = index % (nsamples*ncoils*nrots*nechoes);
    int tmp2 = tmp1 % (nsamples*ncoils*nrots);
    int tmp3 = tmp2 % (nsamples*ncoils);
    int tmp4 = tmp3 % nsamples;

    int sample = tmp4;
    int coil   = (tmp3 - tmp4 ) / nsamples;
    int rot    = (tmp2 - tmp3 ) / (nsamples*ncoils);
    int echoe  = (tmp1 - tmp2 ) / (nsamples*ncoils*nrots);
    int freq   = (index - tmp1) / (nsamples*ncoils*nrots*nechoes);
    
    y_re[index] = 0.0f;
    y_im[index] = 0.0f;

    for( int i=0;i<npix;i++ ){
      
      // EXP
      float angle = -times[sample] * ( fieldmaps[i + rot*npix] + freqs[freq]  )   +    PE_field[ i + rot*npix ];
      float cos_angle = cosf(angle);
      float sin_angle = sinf(angle);
      
      // B1-
      float b1m_re_ = b1m_re[ i + coil*npix + rot*npix*ncoils ];
      float b1m_im_ = b1m_im[ i + coil*npix + rot*npix*ncoils ];
      
      // B1+
      float b1p_re_ = b1p_re[ i + rot*npix ];
      float b1p_im_ = b1p_im[ i + rot*npix ];

      angle = atan2f( b1p_im_,b1p_re_ ) * b1p_echoe_exp[echoe];
      float mag = sqrtf( b1p_re_*b1p_re_ +  b1p_im_*b1p_im_ );

      b1p_re_ = mag * cosf(angle);
      b1p_im_ = mag * sinf(angle);
      

      // compute matrix element for this 
      float A,B,C,D,E,F;
      multiply_2complex( cos_angle,sin_angle,
			 b1m_re_,b1m_im_,
			 &A,&B);

      multiply_2complex( A,B,
			 b1p_re_,b1p_im_,
			 &C,&D);      

      // matrix multiplication
      multiply_2complex( C,D, 
			 x_re[i],x_im[i],
			 &E,&F );

      y_re[index] += E * freq_weights[freq];
      y_im[index] += F * freq_weights[freq];

      
    }
  

  }

}

































