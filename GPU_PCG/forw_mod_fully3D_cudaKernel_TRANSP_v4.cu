
#include <hip/hip_runtime.h>




// this version of the kernel supports PE
// this is the pixel dependent field variation from rot to rot expressed in radians



__device__ void multiply_2complex( float a_re,float a_im,
				   float b_re,float b_im,
				   float *c_re,float *c_im )
{
  *c_re = a_re*b_re - a_im*b_im;
  *c_im = a_re*b_im + a_im*b_re;

}




__global__ void forw_mod_fully3D_cudaKernel_TRANSP_v1( float *x_re,float *x_im,
						       float *y_re,float *y_im,
						       float *times,float *fieldmaps,
						       float *b1m_re,float *b1m_im,
						       float *b1p_re,float *b1p_im,
						       float *b1p_echoe_exp,
						       float *freqs,float *freq_weights,
                               float *PE_field,
                               int nfreqs,
						       int nsamples,int ncoils,int nrots,
						       int nechoes,int npix )
{
  
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  if( index < npix ){
    
    x_re[index] = 0.0f;
    x_im[index] = 0.0f;

    for( int freq=0;freq<nfreqs;freq++ ){

      for( int sample=0;sample<nsamples;sample++ ){
	
	for( int rot=0;rot<nrots;rot++ ){
	  
	  // EXP
	  float angle = -times[sample] * ( fieldmaps[index + rot*npix] + freqs[freq] )   +    PE_field[ index + rot*npix ];;
	  float cos_angle = cosf(angle);
	  float sin_angle = sinf(angle);
	  
	  for( int coil=0;coil<ncoils;coil++ ){
	    
	    // B1-
	    float b1m_re_ = b1m_re[ index + coil*npix + rot*npix*ncoils ];
	    float b1m_im_ = b1m_im[ index + coil*npix + rot*npix*ncoils ];
	    
	    for( int echoe=0;echoe<nechoes;echoe++ ){      
	      
	      // B1+
	      float b1p_re_ = b1p_re[ index + rot*npix ];
	      float b1p_im_ = b1p_im[ index + rot*npix ];

	      angle = atan2f( b1p_im_,b1p_re_ ) * b1p_echoe_exp[echoe];
	      float mag = sqrtf( b1p_re_*b1p_re_ +  b1p_im_*b1p_im_ );

	      b1p_re_ = mag * cosf(angle);
	      b1p_im_ = mag * sinf(angle);
	      
	      // compute matrix element for this 
	      float A,B,C,D,E,F;
	      multiply_2complex( cos_angle,sin_angle,
				 b1m_re_,b1m_im_,
				 &A,&B);
	      
	      multiply_2complex( A,B,
				 b1p_re_,b1p_im_,
				 &C,&D);      
	      
	      // matrix multiplication
	      int data_ind = sample + coil*nsamples + rot*nsamples*ncoils + echoe*nsamples*ncoils*nrots + freq*nsamples*ncoils*nrots*nechoes;
	      
	      multiply_2complex( C,-D, 
				 y_re[data_ind],y_im[data_ind],
				 &E,&F );
	      
	      x_re[index] += E * freq_weights[freq];
	      x_im[index] += F * freq_weights[freq];  

	    }

	  }

	}

      }
      
    }
    
    
  }

}

































